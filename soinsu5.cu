
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>
#include <time.h>

#define target 2*3*5*10000000
#define SIZE 100

double get_cputime(void)
{ 
 struct timespec t;
 clock_gettime(CLOCK_REALTIME,&t);
 //clock_gettime(CLOCK_THREAD_CPUTIME_ID,&t);
 return t.tv_sec + (double)t.tv_nsec*1e-9;
}
double get_realtime(void)
{
 struct timespec t;
 clock_gettime(CLOCK_REALTIME,&t);
 return t.tv_sec + (double)t.tv_nsec*1e-9;
}
double get_tick(void){ return (double)1e-9; }

__host__ int GCD(int a, int b)
{
	int c;
	if(a == 0){
		return b;
	} else {
		c = b % a;
		return GCD(c, a);
	}
}

__global__ void kernel(int *A, int *d_B)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int a = i - j;
	if(i >= __powf(*A,0.5) + 1 && j >= __powf(*A,0.5) + 1 && a > 1 && i < *A && j < *A){
		if(i^2 % *A == j^2 % *A){
			d_B[sizeof(d_B) / sizeof(int)] = a;
		}
	}
}

int main(){
	double t1, t2;
	t1 = get_realtime();
    int *d_target, A = target;
	int *d_B;
	int B[SIZE];
	int i, j, k;
    hipMalloc((void**)&d_target,sizeof(int));
	hipMalloc((void**)&d_B,sizeof(int)*SIZE);
	hipMemcpy(d_target,&A,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_B,&B,sizeof(int)*SIZE,hipMemcpyHostToDevice);
	dim3 block(32,32);
	dim3 grid((A+31)/32,(A+31)/32);
	kernel<<<grid,block>>>(d_target,d_B);
	hipMemcpy(&B,d_B,sizeof(int)*SIZE,hipMemcpyDeviceToHost);
	hipFree(d_target);
	hipFree(d_B);	
	for(i=0;i<SIZE;i++){
		B[i] = GCD(B[i], A);
	}
	B[0] = -1;
	for(i=0;i<SIZE;i++){
		for(k=2;sqrtf(B[i])>=k;k++){
			if(B[i] % k == 0){
				B[i] = 0;
			}
		}
	}
	for(i=0;i<SIZE;i++){
		for(j=i+1;j<SIZE;j++){
			if(B[i] == B[j]){
				B[j] = 0;
			}
		}
	}
	for(i=0;i<SIZE;i++){
		if(B[i] > 1){
			printf("%d ", B[i]);
		}
	}
	printf("\n");
	t2 = get_realtime();
    printf("%10.100f\n", (double)(t2 - t1));
return 0;
}