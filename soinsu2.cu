
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <math.h>

#define target 2*3*5*10000
#define SIZE 100

__device__ int GCD(int *a, int *b)
{
	int c;
	if(*a == 0){
		return *b;
	} else {
		c = *b % *a;
		return GCD(&c, a);
	}
}

__global__ void kernel(int *A, int *d_B, int *d_count)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	int k;
	int a = i - j, b, flag = 0;
	if(i >= __powf(*A,0.5) + 1 && j >= __powf(*A,0.5) + 1 && a > 1 && i < *A && j < *A){
		if(i^2 % *A == j^2 % *A){
			b = GCD(&a, A);
			for(k=2;b>k;k++){
				if(b % k == 0){
					flag = 1;
				}
			}
			if(flag == 0 && b != 1 && *d_count < SIZE){
				d_B[*d_count] = b;
				*d_count = *d_count + 1;
			}
		}
	}
}

int main(){
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
    int *d_target, A = target, count = 0, *d_count;
	int *d_B;
	int B[SIZE];
	int i, j;
	for(i=0;i<SIZE;i++){
		B[i] = 0;
	}
    hipMalloc((void**)&d_target,sizeof(int));
	hipMalloc((void**)&d_B,sizeof(int)*SIZE);
	hipMalloc((void**)&d_count,sizeof(int));
	hipMemcpy(d_target,&A,sizeof(int),hipMemcpyHostToDevice);
	hipMemcpy(d_B,&B,sizeof(int)*SIZE,hipMemcpyHostToDevice);
	hipMemcpy(d_count,&count,sizeof(int),hipMemcpyHostToDevice);
	dim3 block(32,32);
	dim3 grid((A+31)/32,(A+31)/32);
	hipEventRecord(start);
	kernel<<<grid,block>>>(d_target,d_B,d_count);
	hipEventRecord(stop);
	hipEventSynchronize(stop);
	float milliseconds = 0;
	hipEventElapsedTime(&milliseconds, start, stop);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipMemcpy(&B,d_B,sizeof(int)*SIZE,hipMemcpyDeviceToHost);
	hipMemcpy(&count,d_count,sizeof(int),hipMemcpyDeviceToHost);
	hipFree(d_target);
	hipFree(d_B);	
	hipFree(d_count);
	for(i=0;i<SIZE;i++){
		for(j=i+1;j<SIZE;j++){
			if(B[i] == B[j]){
				B[j] = 0;
			}
		}
	}
	for(i=0;i<SIZE;i++){
		if(B[i] != 0){
			printf("%d ", B[i]);
		}
	}
	printf("\n");
	printf("%10.10f\n", milliseconds);
return 0;
}